#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "myProto.h"

// This function is a special - runs on Host and Device
__device__  __host__ double f(int index) {
	int j;
	double value;
	double result = 0;

	for (j = 1; j < HEAVY; j++) {
		value = (index+1)*(j%10);
		result += cos(value);
	}
	return cos(result);
}

__global__ void kernel(int *data, double *arr, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
        arr[i] = f(data[i]);
}


int computeOnGPU(int *data, int numElements, int *result) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    
    size_t size = numElements * sizeof(int);
    
    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory for array of f(data[i]) on device and on host
    double *d_B;
    size = numElements * sizeof(double);
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    double *B = (double *) malloc(size);
    if (B == NULL) {
        fprintf(stderr, "Failed to allocate host memory\n");
        exit(EXIT_FAILURE);
    }
    
    // Launch the Kernel to calculate f(A[i])
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Calculate the result
    *result = 0;
    for (int i = 0;   i < numElements;  i++)
    	if (B[i] > 0)
    		(*result)++;
    		
    // Free allocated memory 
    if (hipFree(d_A) != hipSuccess || hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    free(B);

    return 0;
}

