#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__  void incrementByOne(int *arr, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Increment the proper value of the arrray according to thread ID 
    if (i < numElements)
        arr[i]++;
}


int computeOnGPU(int *data, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);
  

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    incrementByOne<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(data, d_A, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

